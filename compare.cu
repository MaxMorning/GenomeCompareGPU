#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include <fstream>
#include <string>
#include <cstring>
#include <iostream>
#include <algorithm>
#include <vector>
#include <iomanip>
#include "common.h"

#define TILE_SIZE 32

using namespace std;

__constant__ unsigned int compare_seq_length;
__constant__ char compare_seq_genome[MAX_SEQUENCE_LENGTH];

// 计算两条DNA的 Levenshtein 距离
// 通常 str_1 为待比对的DNA，str_2为reference sequence
__device__ int LevenshteinDistance(unsigned int str_2_length, char* str_2)
{
    int buffer_line_0[MAX_SEQUENCE_LENGTH];
    int buffer_line_1[MAX_SEQUENCE_LENGTH];

    int* buffer_line_first = buffer_line_0;
    int* buffer_line_second = buffer_line_1;

    // init first line
    for (int i = 0; i <= compare_seq_length; ++i) {
        buffer_line_first[i] = i;
    }

    for (int line_idx = 1; line_idx <= str_2_length; ++line_idx) {
        buffer_line_second[0] = line_idx;

        for (int i = 1; i <= compare_seq_length; ++i) {
            int replace_delta = 1;
            if (compare_seq_genome[i - 1] == str_2[line_idx - 1]) {
                replace_delta = 0;
            }

            int temp_min = min(min(buffer_line_first[i - 1] + replace_delta, buffer_line_first[i] + 1), buffer_line_second[i - 1] + 1);
            buffer_line_second[i] = temp_min;
        }

        // swap two buffer
        int* temp_buffer_ptr = buffer_line_first;
        buffer_line_first = buffer_line_second;
        buffer_line_second = temp_buffer_ptr;
    }

    int ret_value = buffer_line_first[compare_seq_length];

    return ret_value;
}

// 核函数
// 目前有 1367 条DNA
// 每个block有32 * 32 线程束
__global__ void kernel_func(unsigned int* ref_seq_length_array, char* ref_seq_genome_array, int* distant_result, float* simularity_result) {
    int my_work_seq = threadIdx.x + threadIdx.y * blockDim.x + blockDim.x * blockDim.y * blockIdx.x;
    
    if (my_work_seq >= SEQUENCE_CNT) {
        return;
    }

    char* ref_seq = ref_seq_genome_array + my_work_seq * MAX_SEQUENCE_LENGTH;
    int ref_seq_length = ref_seq_length_array[my_work_seq];
    int distance = LevenshteinDistance(ref_seq_length, ref_seq);
    
    distant_result[my_work_seq] = distance;
    simularity_result[my_work_seq] = 100 - distance * 100.0f / max(ref_seq_length, compare_seq_length);
}

__host__ void load_data_from_disk(unsigned int* h_ref_seq_length_array, char* h_ref_seq_genome_array) {
    ifstream length_file{"length.data", ios::in};
    for (int i = 0; i < SEQUENCE_CNT; ++i) {
        length_file >> h_ref_seq_length_array[i];
    }
    length_file.close();


    FILE* seq_file = fopen("seq.data", "r");
    fread(h_ref_seq_genome_array, MAX_SEQUENCE_LENGTH, SEQUENCE_CNT, seq_file);
    fclose(seq_file);
}

__host__ void load_compare_seq(const string& path) {
    ifstream compare_file{path, ios::in};

    char buffer[MAX_SEQUENCE_LENGTH];
    memset(buffer, 0, MAX_SEQUENCE_LENGTH);

    string buffer_str;
    // 去除首行
    getline(compare_file, buffer_str);

    int char_cnt = 0;
    while (getline(compare_file, buffer_str)) {
        memcpy(buffer + char_cnt, buffer_str.c_str(), buffer_str.length());
        char_cnt += buffer_str.length();
    }

    unsigned int length = strlen(buffer);

    hipMemcpyToSymbol(HIP_SYMBOL(compare_seq_length), &length, sizeof(unsigned int));
    hipMemcpyToSymbol(HIP_SYMBOL(compare_seq_genome), buffer, length);

    compare_file.close();
}

struct SortableItem {
    int distant;
    float simularity;
    string name;

    bool operator<(const SortableItem& item) const {
        return this->distant < item.distant;
    }
};

__host__ void store_result_to_file(const string& path, int* distant_result, float* simularity_result) {
    ifstream seq_name_file{"seq_name.txt", ios::in};
    ofstream result_file{path, ios::out};
    
    string name;

    vector<SortableItem> result_vector;

    for (int i = 0; i < SEQUENCE_CNT; ++i) {
        getline(seq_name_file, name);
        result_vector.push_back({distant_result[i], simularity_result[i], name});
    }

    sort(result_vector.begin(), result_vector.end());

    for (int i = 0; i < SEQUENCE_CNT; ++i) {
        result_file << setw(16) << result_vector[i].name << setw(8) << result_vector[i].distant << setw(16) << result_vector[i].simularity << endl;
    }

    seq_name_file.close();
    result_file.close();
}

int main() {
    dim3 grid{(SEQUENCE_CNT - 1) / (TILE_SIZE * TILE_SIZE) + 1, 1, 1};
    dim3 block{TILE_SIZE, TILE_SIZE, 1};

    unsigned int* h_ref_seq_length_array = new unsigned int[SEQUENCE_CNT];
    char* h_ref_seq_genome_array = new char[SEQUENCE_CNT * MAX_SEQUENCE_LENGTH];

    load_data_from_disk(h_ref_seq_length_array, h_ref_seq_genome_array);


    unsigned int* d_ref_seq_length_array;
    hipMalloc(&d_ref_seq_length_array, SEQUENCE_CNT * sizeof(unsigned int));

    char* d_ref_seq_genome_array;
    hipMalloc(&d_ref_seq_genome_array, SEQUENCE_CNT * MAX_SEQUENCE_LENGTH * sizeof(char));

    hipMemcpy(d_ref_seq_length_array, h_ref_seq_length_array, SEQUENCE_CNT * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_ref_seq_genome_array, h_ref_seq_genome_array, SEQUENCE_CNT * MAX_SEQUENCE_LENGTH * sizeof(char), hipMemcpyHostToDevice);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
    }
    else {
        cout << "Copy ref data to device." << endl;
    }

    // 读取待比较序列
    load_compare_seq("sequences.fasta");
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "Load compare sequence failed: %s\n", hipGetErrorString(cudaStatus));
    }
    else {
        cout << "Load compare data to device." << endl;
    }

    // 为返回值分配显存
    int* d_distant_result;
    hipMalloc(&d_distant_result, SEQUENCE_CNT * sizeof(int));

    float* d_simularity_result;
    hipMalloc(&d_simularity_result, SEQUENCE_CNT * sizeof(float));

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "Alloc result array failed: %s\n", hipGetErrorString(cudaStatus));
    }
    else {
        cout << "Alloc result array in device." << endl;
    }

    cout << "Start calculating..." << endl;
    kernel_func <<<grid, block>>> (d_ref_seq_length_array, d_ref_seq_genome_array, d_distant_result, d_simularity_result);

    hipDeviceSynchronize();

    cout << "Calc done." << endl;

    int h_distant_result[SEQUENCE_CNT];
    float h_simularity_result[SEQUENCE_CNT];

    hipMemcpy(h_distant_result, d_distant_result, SEQUENCE_CNT * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_simularity_result, d_simularity_result, SEQUENCE_CNT * sizeof(float), hipMemcpyDeviceToHost);
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "copy result to host failed: %s\n", hipGetErrorString(cudaStatus));
    }
    else {
        cout << "Copy result to host." << endl;
    }
    // 存回磁盘
    store_result_to_file("result.txt", h_distant_result, h_simularity_result);

    hipFree(d_simularity_result);
    hipFree(d_distant_result);

    hipFree(d_ref_seq_genome_array);
    hipFree(d_ref_seq_length_array);

    delete[] h_ref_seq_genome_array;
    delete[] h_ref_seq_length_array;
    return 0;
}